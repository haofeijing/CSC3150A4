﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	/* Implement open operation here */

	// get file name
	//int i = 0;
	//char tmp = *s;
	//while (tmp != '\0') {
	//	printf("%c", tmp);
	//	i++;
	//	tmp = *(s + i);
	//}
	//printf("\n");

	// find existed file
	for (int i = fs->SUPERBLOCK_SIZE; i < fs->SUPERBLOCK_SIZE + fs->FCB_ENTRIES * fs->FCB_SIZE; i + fs->FCB_SIZE) {
		char tmp = *s;
		int j = 0;
		bool flag = true;
		while (tmp != '\0') {
			if (fs->volume[i + j] != tmp) {
				flag = false;
				break;
			}
			else
			{
				j++;
				tmp = *(s + j);
			}
		}
		if (flag == false) {
			continue;
		}
		else
		{
			return fs->volume[i + j + 1]; // return pointer
		}
	}
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
}
__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
}
